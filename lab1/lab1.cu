#include "hip/hip_runtime.h"
#include "lab1.h"
#include <iostream>
#include <hip/hip_complex.h>
static const unsigned DIMX = 1000;
static const unsigned DIMY = 1000;
static const int BLOCKX = 32;
static const int BLOCKY = 32;
static const unsigned NFRAME = 500;
using namespace std ;
#define DIM 1000

struct Lab1VideoGenerator::Impl {
    int iter=0;
};

Lab1VideoGenerator::Lab1VideoGenerator(): impl(new Impl) {
}

Lab1VideoGenerator::~Lab1VideoGenerator() {}

void Lab1VideoGenerator::get_info(Lab1VideoInfo &info) {
	info.w = DIMX;
	info.h = DIMY;
	info.n_frame = NFRAME;
	info.fps_n = 10;
	info.fps_d = 1;
};


__device__ hipDoubleComplex my_complex_exp(hipDoubleComplex arg)
{
    hipDoubleComplex res;
    double s, c;
    double e = exp(arg.x);
    sincos(arg.y, &s, &c);
    res.x = c * e;
    res.y = s * e;
    return res;
}


__device__ void RGBtoYUV(float *RGBcolor, float *YUVcolor){
    
    YUVcolor[0] = (0.229 * RGBcolor[0]) + (0.587 * RGBcolor[1]) + (0.114 * RGBcolor[2]);
    YUVcolor[1] = -(0.169 * RGBcolor[0]) - (0.331 * RGBcolor[1]) + (0.500 * RGBcolor[2]) + 128;
    YUVcolor[2] =  (0.500 * RGBcolor[0]) - (0.419 * RGBcolor[1]) - (0.081 * RGBcolor[2]) + 128;
}

__global__ void kernel(uint8_t *yuv,hipDoubleComplex c,double zoom,int iTime) {

    hipDoubleComplex ec = my_complex_exp(c);
    ec = hipCmul(make_hipDoubleComplex(0.7885,0),ec);
    int maxiter=256;
    // map from blockIdx to pixel position
    int x =  blockIdx.x * blockDim.x + threadIdx.x;
    int y =  blockIdx.y * blockDim.y + threadIdx.y;
    //int y1 = DIM-y;

    
    if(x<DIM && y<DIM){
        double newRe = 1.5*(x - DIM / 2) / (0.5 * zoom * DIM);
        double newIm = 1.5*(y - DIM/ 2) / (0.5 * zoom * DIM);
        int it;
        
        hipDoubleComplex z = make_hipDoubleComplex(newRe, newIm);
        double smoothcolor = exp(-hipCabs(z));
        for(it = 0; it <maxiter; it++)
        {
            hipDoubleComplex temp =hipCmul(z,z);
            z =hipCmul(z,temp);
            z = hipCadd(z,ec);
            if(hipCabs(z)>4) break;
        }
        
        float  RGBcolor[3];
        float YUVcolor[3];
        if(it==maxiter){
                RGBcolor[2]= 0;
                RGBcolor[1]= 0;
                RGBcolor[0] = 0;
                RGBtoYUV(RGBcolor,YUVcolor);
                yuv[y*DIM+x]=YUVcolor[0];
                if(x%2==0 && y%2==0){
                    yuv[DIM*DIM+(y*DIM)/4+x/2]= YUVcolor[1];
                    yuv[DIM*DIM+DIM*DIM/4+y*DIM/4+x/2]= YUVcolor[2]; 
                }           
        }
        if(it==0){
                RGBcolor[2]= 0;
                RGBcolor[1]= 255;
                RGBcolor[0] = 255;
                RGBtoYUV(RGBcolor,YUVcolor);
                yuv[y*DIM+x]=YUVcolor[0];
                if(x%2==0 && y%2==0){
                    yuv[DIM*DIM+(y*DIM)/4+x/2]= YUVcolor[1];
                    yuv[DIM*DIM+DIM*DIM/4+y*DIM/4+x/2]= YUVcolor[2]; 
                }
        }else if(it==1){
                RGBcolor[2]= 0;
                RGBcolor[1]= 203;
                RGBcolor[0] = 136;
                RGBtoYUV(RGBcolor,YUVcolor);
                yuv[y*DIM+x]=YUVcolor[0];
                if(x%2==0 && y%2==0){
                    yuv[DIM*DIM+(y*DIM)/4+x/2]= YUVcolor[1];
                    yuv[DIM*DIM+DIM*DIM/4+y*DIM/4+x/2]= YUVcolor[2]; 
                }
        }else if(it==2){
                RGBcolor[2]= 0;
                RGBcolor[1]= 128;
                RGBcolor[0] = 0;
                RGBtoYUV(RGBcolor,YUVcolor);
                yuv[y*DIM+x]=YUVcolor[0];
                if(x%2==0 && y%2==0){
                    yuv[DIM*DIM+(y*DIM)/4+x/2]= YUVcolor[1];
                    yuv[DIM*DIM+DIM*DIM/4+y*DIM/4+x/2]= YUVcolor[2]; 
                }
        }else if(it<=4){
                RGBcolor[2]= 0;
                RGBcolor[1]= 255;
                RGBcolor[0] = 0;
                RGBtoYUV(RGBcolor,YUVcolor);
                yuv[y*DIM+x]=YUVcolor[0];
                if(x%2==0 && y%2==0){
                    yuv[DIM*DIM+(y*DIM)/4+x/2]= YUVcolor[1];
                    yuv[DIM*DIM+DIM*DIM/4+y*DIM/4+x/2]= YUVcolor[2]; 
                }
        }else if(it<=8){
                RGBcolor[2]= 255;
                RGBcolor[1]= 255;
                RGBcolor[0] = 255;
                RGBtoYUV(RGBcolor,YUVcolor);
                yuv[y*DIM+x]=YUVcolor[0];
                if(x%2==0 && y%2==0){
                    yuv[DIM*DIM+(y*DIM)/4+x/2]= YUVcolor[1];
                    yuv[DIM*DIM+DIM*DIM/4+y*DIM/4+x/2]= YUVcolor[2]; 
                }
        }else if(it<=16){
                RGBcolor[2]= 256;
                RGBcolor[1]= 149;
                RGBcolor[0] = 245;
                RGBtoYUV(RGBcolor,YUVcolor);
                yuv[y*DIM+x]=YUVcolor[0];
                if(x%2==0 && y%2==0){
                    yuv[DIM*DIM+(y*DIM)/4+x/2]= YUVcolor[1];
                    yuv[DIM*DIM+DIM*DIM/4+y*DIM/4+x/2]= YUVcolor[2]; 
                }
        }else if(it< maxiter/4){
                RGBcolor[2]= 128;
                RGBcolor[1]= 0;
                RGBcolor[0] = 128;
                RGBtoYUV(RGBcolor,YUVcolor);
                yuv[y*DIM+x]=YUVcolor[0];
                if(x%2==0 && y%2==0){
                    yuv[DIM*DIM+(y*DIM)/4+x/2]= YUVcolor[1];
                    yuv[DIM*DIM+DIM*DIM/4+y*DIM/4+x/2]= YUVcolor[2]; 
                }
        }else if(it< maxiter/2){
                RGBcolor[2]= 128;
                RGBcolor[1]= 0;
                RGBcolor[0] = 0;
                RGBtoYUV(RGBcolor,YUVcolor);
                yuv[y*DIM+x]=YUVcolor[0];
                if(x%2==0 && y%2==0){
                    yuv[DIM*DIM+(y*DIM)/4+x/2]= YUVcolor[1];
                    yuv[DIM*DIM+DIM*DIM/4+y*DIM/4+x/2]= YUVcolor[2]; 
                }
        }else if(it< maxiter){
                RGBcolor[2]= 0;
                RGBcolor[1]= 0;
                RGBcolor[0] = 255;
                RGBtoYUV(RGBcolor,YUVcolor);
                yuv[y*DIM+x]=YUVcolor[0];
                if(x%2==0 && y%2==0){
                    yuv[DIM*DIM+(y*DIM)/4+x/2]= YUVcolor[1];
                   yuv[DIM*DIM+DIM*DIM/4+y*DIM/4+x/2]= YUVcolor[2]; 
                }
        }

    }
        
}

int divup(int x, int y) { return x / y + (x % y ? 1 : 0); }

void Lab1VideoGenerator::Generate(uint8_t *yuv) {

    hipMemset(yuv,255, DIM*DIM);
    hipMemset(yuv+DIM*DIM, 128, DIM*DIM/2);
    //double cRe=-0.7;
    double cIm=0.5;
    double zoom=1;
    zoom = zoom + impl->iter * (0.001);

    //cRe = cRe+impl->iter*(0.000003);
    cIm = cIm+impl->iter*(0.012566);
    
    hipDoubleComplex c = make_hipDoubleComplex(0, cIm);
    
    dim3 bs(BLOCKX,BLOCKY);
    dim3 gs(divup(DIM, bs.x), divup(DIM, bs.y));
    kernel <<<gs,bs>>>(yuv,c,zoom,impl->iter);

    ++impl->iter;
}
